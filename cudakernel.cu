
#include <hip/hip_runtime.h>
__global__ void CUDAKernelName(double* C, double* A, double* B, size_t height1, size_t height2, size_t width, double Alpha, double Beta, size_t pitch, size_t offset)
{
	for (int j = blockIdx.y * blockDim.y + threadIdx.y;j < height2;j += blockDim.y * gridDim.y)
	{
		for (int i = blockIdx.x * blockDim.x + threadIdx.x;i < height1;i += blockDim.x * gridDim.x)
		{
			double addval = 0;
#ifdef CALDGEMM_FORCE_K
			for (int k = 0;k < CALDGEMM_FORCE_K;k++)
#else
			for (int k = 0;k < width;k++)
#endif
			{
				addval += A[j * width + k] * B[i * width + k];
			}
			double* destptr = &C[offset + j * pitch + i];
			*destptr = Alpha * addval + Beta * *destptr;
		}
	}
}
